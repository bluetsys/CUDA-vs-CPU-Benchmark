#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA 에러 확인 매크로
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CUDA 커널: 벡터 더하기
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

// C 인터페이스 - 최적화된 버전
extern "C" {
    // 장치 메모리에서 계산하고 결과를 반환
    void cuda_vector_add(const float* h_A, const float* h_B, float* h_C, int N) {
        float* d_A = nullptr;
        float* d_B = nullptr;
        float* d_C = nullptr;
        
        size_t bytes = N * sizeof(float);
        
        // 장치 메모리 할당
        cudaCheck(hipMalloc(&d_A, bytes));
        cudaCheck(hipMalloc(&d_B, bytes));
        cudaCheck(hipMalloc(&d_C, bytes));
        
        // 호스트에서 장치로 데이터 복사 (비동기)
        cudaCheck(hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice));
        cudaCheck(hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice));
        
        // 커널 실행 - 최적화된 블록 크기
        int blockSize = 512;  // 더 큰 블록 크기
        int gridSize = (N + blockSize - 1) / blockSize;
        vectorAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
        cudaCheck(hipGetLastError());
        
        // 장치에서 호스트로 결과 복사 (비동기)
        cudaCheck(hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost));
        
        // 모든 작업 완료 대기
        cudaCheck(hipDeviceSynchronize());
        
        // 메모리 해제
        cudaCheck(hipFree(d_A));
        cudaCheck(hipFree(d_B));
        cudaCheck(hipFree(d_C));
    }
}